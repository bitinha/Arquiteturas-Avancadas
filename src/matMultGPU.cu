
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

using namespace std;

hipEvent_t start, stop;

void startKernelTime (void) {
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
}

void stopKernelTime (void) {
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << milliseconds << " ms have elapsed for the CUDA execution" << endl;
}

void checkCUDAError (const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
		exit(-1);
	}
}

void printMatrix(int N, char c, float *M) {
    cout << c << endl;
    for (int i = 0; i < N; i++) {
        cout << endl;
        for (int j = 0; j < N; j++) {
            cout << M[i * N + j] << "    ";
        }
    }
    cout << "\n\n";
}

void fillMatrices(float *A, float *B, int N) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            A[i*N+j] = ((float)rand()) / ((float)RAND_MAX);
            B[i*N+j] = 1;
        }
    }

}

__global__ void matMultKernel_ijk(int N, float *A, float *B, float *C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < N) {
        C[i * N + j] = 0;
        for (int k = 0; k < N; k++) {
            C[i * N + j] += A[i * N + k] * B[k * N + j];
        }
    }
}

__global__ void matMultKernel_ikj(int N, float *A, float *B, float *C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && k < N/*  && k >= 1 */) {
        for (int j = 0; j < N; j++) {
            C[i * N + j] += A[i * N + k] * B[k * N + j];
        }
    }

    // void matMult_ikj(float *A, float *B, float *C, int N) {
    //     for (int i = 0; i < N; i++) {
    //         for (int j = 0; j < N; j++) {   //1ª iteração é necessário definir ter a matriz resultado a 0
    //             C[i*N+j] = A[i*N+0] * B[0*N+j];
    //         }
    //         for (int k = 1; k < N; k++) {
    //             for (int j = 0; j < N; j++) {
    //                 C[i*N+j] += A[i*N+k] * B[k*N+j];
    //             }
    //         }
    //     }
    // }
}

void matMultGPU(int N, float *A, float *B, float *C) {
    dim3 threadsPerBlock(8, 8);
    dim3 blocksPerGrid(2, 2);

    // declare variable with size of the array in bytes
    int bytes = N * N * sizeof(float);

	// pointers to the device memory
    float *dA, *dB, *dC; 

	// allocate the memory on the device
	hipMalloc((void**) &dA, bytes);
	hipMalloc((void**) &dB, bytes);
	hipMalloc((void**) &dC, bytes);
    
    startKernelTime();

    checkCUDAError("mem allocation");
	// copy inputs to the device
	hipMemcpy(dA, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, bytes, hipMemcpyHostToDevice);


    hipMemcpy(dC, C, bytes, hipMemcpyHostToDevice); /** Temporário **/

	checkCUDAError("memcpy h->d");
    // launch the kernel
	matMultKernel_ikj <<< threadsPerBlock, blocksPerGrid >>> (N, dA, dB, dC);

	checkCUDAError("kernel invocation");
	// copy the output to the host
	hipMemcpy(C, dC, bytes, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy d->h");
    
    stopKernelTime();

	// free the device memory
	hipFree(dA); hipFree(dB); hipFree(dC);
	checkCUDAError("mem free");
}

int main(int argc, char const *argv[]) {
    if (argc == 1) {
        cout << "Insira o tamanho da matriz" << endl;
        exit(1);
    }

    int const N = atoi(argv[1]);

    float A[N * N];
    float B[N * N];
    float C[N * N];

    fillMatrices(A, B, N);

    /** Temporário **/
    for (int i = 0; i < N * N; i++) {
        C[i] = 0;
    }

    matMultGPU(N, A, B, C);

    printMatrix(N, 'A', A);
    printMatrix(N, 'B', B);
    printMatrix(N, 'C', C);
}
