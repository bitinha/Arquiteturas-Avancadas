
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

#define NUM_BLOCKS 16
#define NUM_THREADS_PER_BLOCK 16
#define N 16

using namespace std;

hipEvent_t start, stop;

void startKernelTime (void) {
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
}

void stopKernelTime (void) {
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << milliseconds << " ms have elapsed for the CUDA execution" << endl;
}

void checkCUDAError (const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
		exit(-1);
	}
}

void printMatrix(char c, float *M) {
    cout << c << endl;
    for (int i = 0; i < N; i++) {
        cout << endl;
        for (int j = 0; j < N; j++) {
            cout << M[i * N + j] << "    ";
        }
    }
    cout << "\n\n";
}

void fillMatrices(float *A, float *B) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            A[i*N+j] = ((float)rand()) / ((float)RAND_MAX);
            B[i*N+j] = 1;
        }
    }

}

__global__ void matMultKernel_ijk(float *A, float *B, float *C) {
    int i = blockIdx.x;
    int j = threadIdx.x;

    if (i < N && j < N) {
        C[i * N + j] = 0;
        for (int k = 0; k < N; k++) {
            C[i * N + j] += A[i * N + k] * B[k * N + j];
        }
    }
}

__global__ void matMultKernel_ikj(float *A, float *B, float *C) {
    int i = blockIdx.x;
    int k = threadIdx.x;

    if (i < N && k < N/*  && k >= 1 */) {
        for (int j = 0; j < N; j++) {
            C[i * N + j] += A[i * N + k] * B[k * N + j];
        }
    }

    // void matMult_ikj(float *A, float *B, float *C, int N) {
    //     for (int i = 0; i < N; i++) {
    //         for (int j = 0; j < N; j++) {   //1ª iteração é necessário definir ter a matriz resultado a 0
    //             C[i*N+j] = A[i*N+0] * B[0*N+j];
    //         }
    //         for (int k = 1; k < N; k++) {
    //             for (int j = 0; j < N; j++) {
    //                 C[i*N+j] += A[i*N+k] * B[k*N+j];
    //             }
    //         }
    //     }
    // }
}

void matMultGPU(float *A, float *B, float *C) {
    // declare variable with size of the array in bytes
    int bytes = N * N * sizeof(float);

	// pointers to the device memory
    float *dA, *dB, *dC; 

	// allocate the memory on the device
	hipMalloc((void**) &dA, bytes);
	hipMalloc((void**) &dB, bytes);
	hipMalloc((void**) &dC, bytes);
    
    startKernelTime();

    checkCUDAError("mem allocation");
	// copy inputs to the device
	hipMemcpy(dA, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, bytes, hipMemcpyHostToDevice);


    //cudaMemcpy(dC, C, bytes, cudaMemcpyHostToDevice); /** Temporário **/

	checkCUDAError("memcpy h->d");
    // launch the kernel
	matMultKernel_ijk <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS >>> (dA, dB, dC);
	// matMultKernel_ikj <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS >>> (dA, dB, dC);

	checkCUDAError("kernel invocation");
	// copy the output to the host
	hipMemcpy(C, dC, bytes, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy d->h");
    
    stopKernelTime();

	// free the device memory
	hipFree(dA); hipFree(dB); hipFree(dC);
	checkCUDAError("mem free");
}

int main(int argc, char const *argv[]) {
    float A[N * N];
    float B[N * N];
    float C[N * N];

    fillMatrices(A, B);

    // /** Temporário **/
    // for (int i = 0; i < N * N; i++) {
    //     C[i] = 0;
    // }

    matMultGPU(A, B, C);

    printMatrix('A', A);
    printMatrix('B', B);
    printMatrix('C', C);
}
