
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <iostream>

using namespace std;

hipEvent_t start, stop;

void startKernelTime (void) {
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
}

void stopKernelTime (void) {
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << milliseconds << " ms have elapsed for the CUDA execution" << endl;
}

void checkCUDAError (const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
		exit(-1);
	}
}

void printMatrix(int N, char c, float *M) {
    printf("%c:\n", c);
    for (int i = 0; i < N; i++) {
        printf("\n");
        for (int j = 0; j < N; j++) {
            printf("%f\t", M[i * N + j]);
        }
    }
    printf("\n\n");
}

//https://en.wikipedia.org/wiki/In-place_matrix_transposition#Square_matrices
void transpose(int N, float *__restrict__ M) {
    float temp;

    for (int i = 0; i < N - 2; ++i) {
        // #pragma omp simd
        for (int j = i + 1; j < N - 1; ++j) {
            temp = M[i * N + j];
            M[i * N + j] = M[j * N + i];
            M[j * N + i] = temp;
        }
    }
}

__global__ void matMultKernel(int N, float *A, float *B, float *C) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && col < N) {
        C[row * N + col] = 0;
        for (int k = 0; k < N; k++) {
            C[row * N + col] += A[row * N + k] * B[col * N + k];
        }
    }
}

void matMultGPU(int N, float *A, float *B, float *C) {
    transpose(N,B);

    dim3 threadsPerBlock(8, 8);
    dim3 blocksPerGrid(4, 4);

    // declare variable with size of the array in bytes
    int bytes = N * N * sizeof(float);

	// pointers to the device memory
    float *dA, *dB, *dC; 

	// allocate the memory on the device
	hipMalloc((void**) &dA, bytes);
	hipMalloc((void**) &dB, bytes);
	hipMalloc((void**) &dC, bytes);
    
    startKernelTime();

    checkCUDAError("mem allocation");
	// copy inputs to the device
	hipMemcpy(dA, A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, bytes, hipMemcpyHostToDevice);

	checkCUDAError("memcpy h->d");
	// launch the kernel
	matMultKernel <<< threadsPerBlock, blocksPerGrid >>> (N, dA, dB, dC);

	checkCUDAError("kernel invocation");
	// copy the output to the host
	hipMemcpy(C, dC, bytes, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy d->h");
    
    stopKernelTime();

	// free the device memory
	hipFree(dA); hipFree(dB); hipFree(dC);
	checkCUDAError("mem free");
}

int main(int argc, char const *argv[]) {
    // if (argc == 1) {
    //     printf("Insira o tamanho da matriz\n");
    //     exit(1);
    // }

    // int const N = atoi(argv[1]);
    int N = 16;

    // float *A = malloc(sizeof(float) * N * N);
    // float *B = malloc(sizeof(float) * N * N);
    // float *C = malloc(sizeof(float) * N * N);
    float A[N * N];
    float B[N * N];
    float C[N * N];

    float r;

    for (int i = 0; i < N; ++i) {
        r = ((float)rand()) / ((float)RAND_MAX);
        for (int j = 0; j < N; ++j) {
            A[i * N + j] = r;
            B[i * N + j] = 1;
        }
    }

    matMultGPU(N, A, B, C);

    printMatrix(N, 'A', A);
    printMatrix(N, 'B', B);
    printMatrix(N, 'C', C);
}
